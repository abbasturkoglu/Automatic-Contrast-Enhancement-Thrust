#include "hip/hip_runtime.h"
// Do not alter the preprocessor directives
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <cstdlib>
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/pair.h"
#include "thrust/extrema.h"


#define NUM_CHANNELS 1

#define MinVal(x, y) (((x) < (y)) ? (x) : (y))
#define MaxVal(x, y) (((x) > (y)) ? (x) : (y))

//This function multiply every pixel value with scale constant
struct multiplyFunction
{
	float a;
 

	multiplyFunction(float s_constant) {
		a = s_constant; 
	}

	__host__ __device__
		uint8_t operator()(const uint8_t& x) const
	{
		return a*x ;
	}
};

//This function is another option to contrast image with one epoch. Subtracts the minimum value and multiply with scale constant together
struct scaleFunction
{
	unsigned int a;
 unsigned int b;

	scaleFunction(unsigned int s_constant,unsigned int min) {
		a = s_constant; 
    b= min;
	}

	__host__ __device__
		uint8_t operator()(const uint8_t& x) const
	{
		return (x-b)*a ;
	}
};

int main() {

	int width; //image width
	int height; //image height
	int bpp;  //bytes per pixel if the image was RGB (not used)


	// Load a grayscale bmp image to an unsigned integer array with its height and weight.
	//  (uint8_t is an alias for "unsigned char")
  uint8_t* image =  stbi_load("./samples/640x426.bmp", &width, &height, &bpp, NUM_CHANNELS);
  size_t image_size = width * height * sizeof(uint8_t);




	// Print for sanity check
	printf("Bytes per pixel: %d \n", bpp / 3); //Image is grayscale, so bpp / 3;
	printf("Height: %d \n", height);
	printf("Width: %d \n", width);


	//Start Counter
	hipEvent_t start, stop;
	float elapsed_time_ms;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


  //Create device vector image_d and initialize with value of image
  thrust::device_vector<uint8_t> image_d(image, image + (width * height));

  // Find minimum and maximum values
  int min_t = thrust::reduce(image_d.begin(), image_d.end(),255, thrust::minimum<int>());
  int max_t = thrust::reduce(image_d.begin(), image_d.end(), 0, thrust::maximum<int>());
  
  
  float scale_constant = 255.0f / (max_t - min_t);

  // I designed 3 different thrust kernels. They do the same job with different methods
  // Their performances are very similar so it does not matter which one you use
  
  //option 1 for subtract and scale
  thrust::for_each(image_d.begin(), image_d.end(), thrust::placeholders::_1 -= min_t);
  thrust::transform(image_d.begin(), image_d.end(), image_d.begin(), multiplyFunction(scale_constant));

  //option 2 for subtract and scale
  //thrust::for_each(image_d.begin(), image_d.end(), thrust::placeholders::_1 -= min_t);
  //thrust::for_each(image_d.begin(), image_d.end(), thrust::placeholders::_1 *= scale_constant);


  //option 3 for subtract and scale
  //thrust::transform(image_d.begin(), image_d.end(), image_d.begin(), scaleFunction(scale_constant,min_t));

  //Copy enhanced image to host
  thrust::host_vector<uint8_t> image_h(image_d);
	uint8_t* image_e = image_h.data();



	//Stop timing
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	printf("\nTime to calculate results(GPU Time): %f ms.\n\n", elapsed_time_ms);


	// Write image array into a bmp file
	stbi_write_bmp("./samples/out_img.bmp", width, height, 1, image_e);
	printf("\nEnchanced image successfully saved.\n\n");

  //print minimum and maximum value
	printf("Minimum Pixel Value: %d\n", min_t);
	printf("Maximum Pixel Value: %d\n", max_t);



	return 0;
}